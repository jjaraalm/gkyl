
#include <hip/hip_runtime.h>
__global__
void sumArray(int n, double a, double *x, double *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i<n) y[i] = a*x[i]+y[i];
}
